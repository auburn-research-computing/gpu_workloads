
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello() { 
   printf("Hello World!\n"); 
}

int main() { 
   cuda_hello<<<6,1>>>();
   hipDeviceSynchronize();
   return 0; 
}
